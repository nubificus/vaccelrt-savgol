#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <sstream>
#include <unistd.h>

#include <stdint.h>
#include <stdlib.h>
#include <time.h>

#include "prometheus/client_metric.h"
#include "prometheus/counter.h"
#include "prometheus/exposer.h"
#include "prometheus/family.h"
#include "prometheus/info.h"
#include "prometheus/registry.h"

#include "prometheus/gauge.h"

#include <cstdio>
#include <cstring>
#include <algorithm>
#include <random>
#include <cstring>
#include <cmath>
#include <chrono>
#include <ctime>

using namespace std;

#define DATA_SIZE 200000
#define window 11
#define polyorder 3
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#define BLOCKSIZE 32

struct timeval t0, t1, t2, t3;
#define DEFAULT_NL (15) // half window
#define DEFAULT_NR (15) // half window
#define DEFAULT_M (4)   // polynomial order
#define DEFAULT_LD (0)  // derivative order
#define EPSILON ((double)(1.0e-20))

__global__ void warming_kernel()
{
    int i = 7;
}

__global__ void savgol_kernel(double *indata, double *c, double *outdata, int mm, int nl, int nr)
{
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    int j;
    if ((index >= 1) && (index <= nl))
    {

        for (outdata[index - 1] = 0.0, j = -nl; j <= nr; j++)
        {
            if (index + j >= 1)
            {

                int dd = index + j - 1;
                outdata[index - 1] += c[(j >= 0 ? j + 1 : nr + nl + 2 + j)] * indata[dd];
            }
        }
    }

    if ((index >= nl + 1) && (index <= mm - nr))
    {
        for (outdata[index - 1] = 0.0, j = -nl; j <= nr; j++)
        {
            outdata[index - 1] += c[(j >= 0 ? j + 1 : nr + nl + 2 + j)] * indata[index + j - 1];
        }
    }

    if ((index >= mm - nr + 1) && (index <= mm))
    {
        for (outdata[index - 1] = 0.0, j = -nl; j <= nr; j++)
        {
            if (index + j <= mm)
            {
                outdata[index - 1] += c[(j >= 0 ? j + 1 : nr + nl + 2 + j)] * indata[index + j - 1];
            }
        }
    }
}

//////////////////////

///////////////////////////////////////////////////////////////////////////

void free_dmatrix(double **m, long nrl, long nrh, long ncl, long nch)
{
    free((char *)(m[nrl] + ncl - 1));
    free((char *)(m + nrl - 1));
}

int *ivector(long nl, long nh)
{
    int *v;
    v = (int *)malloc((size_t)((nh - nl + 2) * sizeof(int)));
    if (!v)
    {
        // log("Error: Allocation failure.");
        exit(1);
    }
    return v - nl + 1;
}

void free_ivector(int *v, long nl, long nh)
{
    free((char *)(v + nl - 1));
}
void free_dvector(double *v, long nl, long nh)
{
    free((char *)(v + nl - 1));
}

double *dvector(long nl, long nh)
{
    double *v;
    long k;
    v = (double *)malloc((size_t)((nh - nl + 2) * sizeof(double)));
    if (!v)
    {
        // log("Error: Allocation failure.");
        exit(1);
    }
    for (k = nl; k <= nh; k++)
        v[k] = 0.0;
    return v - nl + 1;
}

void lubksb(double **a, int n, int *indx, double b[])
{
    int i, ii = 0, ip, j;
    double sum;

    for (i = 1; i <= n; i++)
    {
        ip = indx[i];
        sum = b[ip];
        b[ip] = b[i];
        if (ii)
            for (j = ii; j <= i - 1; j++)
                sum -= a[i][j] * b[j];
        else if (sum)
            ii = i;
        b[i] = sum;
    }
    for (i = n; i >= 1; i--)
    {
        sum = b[i];
        for (j = i + 1; j <= n; j++)
            sum -= a[i][j] * b[j];
        b[i] = sum / a[i][i];
    }
}

double **dmatrix(long nrl, long nrh, long ncl, long nch)
{
    long i, nrow = nrh - nrl + 1, ncol = nch - ncl + 1;
    double **m;
    m = (double **)malloc((size_t)((nrow + 1) * sizeof(double *)));
    if (!m)
    {
        // log("Allocation failure 1 occurred.");
        exit(1);
    }
    m += 1;
    m -= nrl;
    m[nrl] = (double *)malloc((size_t)((nrow * ncol + 1) * sizeof(double)));
    if (!m[nrl])
    {
        // log("Allocation failure 2 occurred.");
        exit(1);
    }
    m[nrl] += 1;
    m[nrl] -= ncl;
    for (i = nrl + 1; i <= nrh; i++)
        m[i] = m[i - 1] + ncol;
    return m;
}

void ludcmp(double **a, int n, int *indx, double *d)
{
    int i, imax = 0, j, k;
    double big, dum, sum, temp;
    double *vv;

    vv = dvector(1, n);
    *d = 1.0;
    for (i = 1; i <= n; i++)
    {
        big = 0.0;
        for (j = 1; j <= n; j++)
            if ((temp = fabs(a[i][j])) > big)
                big = temp;
        if (big == 0.0)
        {
            // log("Error: Singular matrix found in routine ludcmp()");
            exit(1);
        }
        vv[i] = 1.0 / big;
    }
    for (j = 1; j <= n; j++)
    {
        for (i = 1; i < j; i++)
        {
            sum = a[i][j];
            for (k = 1; k < i; k++)
                sum -= a[i][k] * a[k][j];
            a[i][j] = sum;
        }
        big = 0.0;
        for (i = j; i <= n; i++)
        {
            sum = a[i][j];
            for (k = 1; k < j; k++)
                sum -= a[i][k] * a[k][j];
            a[i][j] = sum;
            if ((dum = vv[i] * fabs(sum)) >= big)
            {
                big = dum;
                imax = i;
            }
        }
        if (j != imax)
        {
            for (k = 1; k <= n; k++)
            {
                dum = a[imax][k];
                a[imax][k] = a[j][k];
                a[j][k] = dum;
            }
            *d = -(*d);
            vv[imax] = vv[j];
        }
        indx[j] = imax;
        if (a[j][j] == 0.0)
            a[j][j] = EPSILON;
        if (j != n)
        {
            dum = 1.0 / (a[j][j]);
            for (i = j + 1; i <= n; i++)
                a[i][j] *= dum;
        }
    }
    free_dvector(vv, 1, n);
}

char sgcoeff(double c[], int np, int nl, int nr, int ld, int m)
{
    void lubksb(double **a, int n, int *indx, double b[]);
    void ludcmp(double **a, int n, int *indx, double *d);
    int imj, ipj, j, k, kk, mm, *indx;
    double d, fac, sum, **a, *b;

    if (np < nl + nr + 1 || nl < 0 || nr < 0 || ld > m || nl + nr < m)
    {
        // log("Inconsistent arguments detected in routine sgcoeff.");
        return (1);
    }
    indx = ivector(1, m + 1);
    a = dmatrix(1, m + 1, 1, m + 1);
    b = dvector(1, m + 1);
    for (ipj = 0; ipj <= (m << 1); ipj++)
    {
        sum = (ipj ? 0.0 : 1.0);
        for (k = 1; k <= nr; k++)
            sum += pow((double)k, (double)ipj);
        for (k = 1; k <= nl; k++)
            sum += pow((double)-k, (double)ipj);
        mm = (ipj < 2 * m - ipj ? ipj : 2 * m - ipj);
        for (imj = -mm; imj <= mm; imj += 2)
            a[1 + (ipj + imj) / 2][1 + (ipj - imj) / 2] = sum;
    }
    ludcmp(a, m + 1, indx, &d);
    for (j = 1; j <= m + 1; j++)
        b[j] = 0.0;
    b[ld + 1] = 1.0;
    lubksb(a, m + 1, indx, b);
    for (kk = 1; kk <= np; kk++)
        c[kk] = 0.0;
    for (k = -nl; k <= nr; k++)
    {
        sum = b[1];
        fac = 1.0;
        for (mm = 1; mm <= m; mm++)
            sum += b[mm + 1] * (fac *= k);
        kk = ((np - k) % np) + 1;
        c[kk] = sum;
    }
    free_dvector(b, 1, m + 1);
    free_dmatrix(a, 1, m + 1, 1, m + 1);
    free_ivector(indx, 1, m + 1);
    return (0);
}
/////////////////////////


struct vaccel_arg {
        uint32_t len;
        uint8_t *buf;
};
extern "C" 

int savgol_GPU(int argc, char **argv, double *time1, double *time2)
{

    //---- prometheus stuff ----
    using namespace prometheus;
    // create a http server running on port 8080
    Exposer exposer{"127.0.0.1:8082"};
    // create a metrics registry
    // @note it's the users responsibility to keep the object alive
    auto registry = std::make_shared<Registry>();

    auto &latencys_gauge = BuildGauge()
                               .Name("latencys_gauge")
                               .Help("latency in sec")
                               .Register(*registry);

    auto &latencyms_gauge = BuildGauge()
                                .Name("latencyms_gauge")
                                .Help("latency in ms")
                                .Register(*registry);

    auto &result_gauge = BuildGauge()
                             .Name("result_gauge")
                             .Help("result")
                             .Register(*registry);

    auto &Throughput_gauge = BuildGauge()
                                 .Name("Throughput_gauge")
                                 .Help("Throughput")
                                 .Register(*registry);

    auto &version_info = BuildInfo()
                             .Name("versions")
                             .Help("Static info about the library")
                             .Register(*registry);

    version_info.Add({{"prometheus", "1.0"}});

    std::string FILE_PATH = argv[1];
    string data_file_path = FILE_PATH;
    // string golden_file_path = "dataset.txt";
    string line;

    cout << "* Savgol Filter *" << endl;
    //cout << " # input file:               " << data_file_path << endl;
    // cout << " # golden file:                " << golden_file_path << endl;

    double *indata, *outdata;
    indata = (double *)malloc(DATA_SIZE * sizeof(double));
    outdata = (double *)malloc(DATA_SIZE * sizeof(double));
    int rowcount = DATA_SIZE;

    if (strcmp(argv[0],"vaccel") == 0 ) {
          std::istringstream file(FILE_PATH);
    int index = 0;
    while (getline(file, line)) {
        indata[index] = (float)atof(line.c_str());
        index++;
        //  cout << to_string(data[index-1]) << endl;
    }


    } else {
    cout << " # input file:               " << data_file_path << endl;
    // read input data
    ifstream data_file;
    data_file.open(FILE_PATH);
    int index = 0;
    while (getline(data_file, line))
    {
        indata[index] = (float)atof(line.c_str());
        index++;
        //  cout << to_string(data[index-1]) << endl;
    }
    data_file.close();


  }


    int nl; //= DEFAULT_NL;
    int nr; //= DEFAULT_NR;
    int ld = DEFAULT_LD;
    int m; //= DEFAULT_M;
    int mm = rowcount;

    printf("Parameters=%d %d\n\n", window, polyorder);
    nl = window / 2;
    nr = nl;
    m = polyorder;

    int np = nl + 1 + nr;
    double *c;
    char retval;

    int j;
    long int k;
    c = dvector(1, nl + nr + 1);

    retval = sgcoeff(c, np, nl, nr, ld, m);

    warming_kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    gettimeofday(&t0, NULL);
    if (retval == 0)
    {
        double *indata_gpu;
        double *outdata_gpu;
        double *c_gpu;
        double *x;
        int c_size = nl + nr + 2;
        hipMalloc((void **)&indata_gpu, sizeof(double) * rowcount);
        hipMalloc((void **)&outdata_gpu, sizeof(double) * rowcount);
        hipMalloc((void **)&c_gpu, sizeof(double) * c_size);
        hipMemcpy(indata_gpu, indata, sizeof(double) * rowcount, hipMemcpyHostToDevice);
        hipMemcpy(c_gpu, c, sizeof(double) * c_size, hipMemcpyHostToDevice);
        dim3 block(BLOCKSIZE, 1);
        dim3 grid((size_t)(ceil(((float)rowcount + 1) / ((float)block.x))), 1);
        chrono::high_resolution_clock::time_point tt1, tt2;
        tt1 = chrono::high_resolution_clock::now();
        gettimeofday(&t2, NULL);
        savgol_kernel<<<grid, block>>>(indata_gpu, c_gpu, outdata_gpu, rowcount, nl, nr);
        hipDeviceSynchronize();
        gettimeofday(&t3, NULL);
        tt2 = chrono::high_resolution_clock::now();
        chrono::duration<double> Latency = tt2 - tt1;
        int input_size_in_kbytes = 2 * (rowcount * sizeof(double)) / 1024;

        float Throughput = (float)input_size_in_kbytes / Latency.count();
        // ask the exposer to scrape the registry on incoming HTTP requests
        exposer.RegisterCollectable(registry);
        latencys_gauge.Add({{"gauge", "latency in sec"}}).Set(Latency.count());
        latencyms_gauge.Add({{"gauge", "latency in msec"}}).Set(Latency.count() * 1000);
        Throughput_gauge.Add({{"gauge", "result"}}).Set(Throughput);
#if 0
        for (;;)
        {
            std::cout << "Latency in sec " << Latency.count() << std::endl;
            std::cout << "Latency in msec " << Latency.count() * 1000 << std::endl;
            std::cout << "Throughput (KB/sec): " << Throughput << std::endl;
        }
#endif
        hipMemcpy(outdata, outdata_gpu, sizeof(double) * rowcount, hipMemcpyDeviceToHost);
    }
    gettimeofday(&t1, NULL);
    double t10 = (t1.tv_sec * 1000000.0 + t1.tv_usec) - (t0.tv_sec * 1000000.0 + t0.tv_usec);
    fprintf(stderr, "total GPU process time: %lf msecs\n", (t10) / 1000.0F);
    double t32 = (t3.tv_sec * 1000000.0 + t3.tv_usec) - (t2.tv_sec * 1000000.0 + t2.tv_usec);
    fprintf(stderr, "only savgol GPU kernel time: %lf msecs\n", (t32) / 1000.0F);
    *time1 = t10;
    *time2 = t32;

    free_dvector(c, 1, nr + nl + 1);

    for (int i = 0; i < 10; i++)
    {

        printf("%lf ", indata[i]);
        printf("%lf\n ", outdata[i]);
    }

    ///////////////////////////////////////////////

    return 0;
}

extern "C" 

int savgol_GPU_unpack(void *out_args, size_t out_nargs, void* in_args, size_t in_nargs)
{

	struct vaccel_arg *in_arg = (struct vaccel_arg*)in_args;
        struct vaccel_arg *out_arg = (struct vaccel_arg*)out_args;

        int argc = 2;
	double time1, time2;
        char *argv[2] = {
                "vaccel",
                (char *)out_arg[0].buf
        };

        //printf("argv0=%s, %s\n", argv[0], argv[1]);
        //printf("out_arg[0]=%lf\n", *(float *)out_arg[0].buf);
        int ret = savgol_GPU(argc, argv, &time1, &time2);
        printf("ret=%d time1 %lf, time2 %lf\n", ret, time1, time2);

#if 1
        *(double*)in_arg[0].buf = time1;
        in_arg[0].len = sizeof(double);
        *(double*)in_arg[1].buf = time2;
        in_arg[1].len = sizeof(double);
#endif

//      fflush(stdout);
    	hipDeviceReset();
        return 0;
}


